#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCKS 4
#define THREADS 4

#define LEN 10
#define MAX 10000

__global__ void rand_init(int* x) {
    int block_id = blockIdx.x;
    x[block_id] = rand() % MAX;
} 

__global__ void add_vector(int *x, int *y, int *result) {
    int block_id = blockIdx.x;
    if(block_id < LEN)
        result[block_id] = x[block_id] + y[block_id];
};

void print_arr(int arr[], int len) {
    printf("[");
    for(int i = 0; i < len-1; i++)
        printf("%d, ", arr[i]);
    printf("%d]\n", arr[len-1]);
};

int main(int argc, char *argv[]) {
    int i, x[LEN], y[LEN], sum[LEN];
    int *x_d, *y_d, *sum_d;

    hipMalloc((void**) &x_d, LEN * sizeof(int));
    hipMalloc((void**) &y_d, LEN * sizeof(int));
    hipMalloc((void**) &sum_d, LEN * sizeof(int));
    
    for(i = 0; i < LEN; i++)
    {
        x[i] = rand() % 1000;
        y[i] = rand() % 1000;
    }

    print_arr(x, LEN);
    print_arr(y, LEN);

    hipMemcpy(y_d, &y, LEN * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(x_d, &x, LEN * sizeof(int), hipMemcpyHostToDevice);

    add_vector<<<LEN,1>>>(x_d, y_d, sum_d);


    hipMemcpy(&sum, sum_d, LEN * sizeof(int), hipMemcpyDeviceToHost);

    print_arr(sum, LEN);
    
    hipFree(x_d);
    hipFree(y_d);
    hipFree(sum_d);
};
