#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


#define BLOCKS 4
#define THREADS 4


__global__ void add(int *x, int *y, int *result) {
    *result = *x + *y;
};

int main(int argc, char *argv[]) {
    if(argc < 2)
    {
        printf("need two parameters retard\n");
        return 0;
    }
    int x = atoi(argv[1]);
    int y = atoi(argv[2]);

    int *x_d, *y_d, *sum_d;

    hipMalloc((void**) &x_d, sizeof(int));
    hipMemcpy(x_d, &x, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &y_d, sizeof(int));
    hipMemcpy(y_d, &y, sizeof(int), hipMemcpyHostToDevice);
    hipMalloc((void**) &sum_d, sizeof(int));

    add<<<BLOCKS,THREADS>>>(x_d, y_d, sum_d);

    int sum;
    hipMemcpy(&sum, sum_d, sizeof(int), hipMemcpyDeviceToHost);
    printf("%d\n", sum);
    
    hipFree(x_d);
    hipFree(y_d);
    hipFree(sum_d);
};
