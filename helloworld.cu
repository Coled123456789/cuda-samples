
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int dev1(){
    return 1;
}

__device__ int dev2(){
    return 2;
}

/*
* __global__ prefix says a function is kernel,
*   Will be executed by GPU
*   runs multiple times specified by block and thread number
*   must return void
*/
__global__ void myKernel(){
    dev1();
    dev2();
}


/**
*  __host__ prefix specifies
*   - runs once per call on CPU
*   - only callable from CPU
* Function without prefix are host functions
*/

int main(){
    //specifies number of blocks and threads per blocks (2 blocks, 4 threads per block)
    myKernel<<<2,4>>>();
    printf("Hello, World!\n");
    return 0;
}

/*
*  __device prefix
*/
